/**
 *   ___ _   _ ___   _     _       ___ ___ ___ ___
 *  / __| | | |   \ /_\   | |  ___| _ ) __/ __/ __|
 * | (__| |_| | |) / _ \  | |_|___| _ \ _| (_ \__ \
 *  \___|\___/|___/_/ \_\ |____|  |___/_| \___|___/
 *                                               2012
 *     by Jens Wetzl           (jens.wetzl@fau.de)
 *    and Oliver Taubmann (oliver.taubmann@fau.de)
 *
 * This work is licensed under a Creative Commons
 * Attribution 3.0 Unported License. (CC-BY)
 * http://creativecommons.org/licenses/by/3.0/
 *
 * File cost_function.cu: Implementation of cost function classes.
 *
 **/

#include "cost_function.h"
#include "error_checking.h"

void cpu_cost_function::f_gradf(const float *d_x, float *d_f, float *d_gradf)
{
	// Copy device x to host memory
	CudaSafeCall( hipMemcpy(m_h_x, d_x, m_numDimensions * sizeof(float), hipMemcpyDeviceToHost) );
	float h_f;

#ifdef LBFGS_CPU_DOUBLE_PRECISION
	std::cerr << "Don't try to use the GPU minimizer when LBFGS_CPU_DOUBLE_PRECISION is enabled." << std::endl;
	exit(EXIT_FAILURE);
#else
	cpu_f_gradf(m_h_x, &h_f, m_h_gradf);
#endif

	// Copy host f and gradf to device memory
	CudaSafeCall( hipMemcpy(d_f,         &h_f,                   sizeof(float), hipMemcpyHostToDevice) );
	CudaSafeCall( hipMemcpy(d_gradf, m_h_gradf, m_numDimensions * sizeof(float), hipMemcpyHostToDevice) );
}

#include "hip/hip_runtime.h"
﻿/**
 *   ___ _   _ ___   _     _       ___ ___ ___ ___
 *  / __| | | |   \ /_\   | |  ___| _ ) __/ __/ __|
 * | (__| |_| | |) / _ \  | |_|___| _ \ _| (_ \__ \
 *  \___|\___/|___/_/ \_\ |____|  |___/_| \___|___/
 *                                               2012
 *     by Jens Wetzl           (jens.wetzl@fau.de)
 *    and Oliver Taubmann (oliver.taubmann@fau.de)
 *
 * This work is licensed under a Creative Commons
 * Attribution 3.0 Unported License. (CC-BY)
 * http://creativecommons.org/licenses/by/3.0/
 *
 * File lbfgs.cu: Implementation of class lbfgs (except cpu_lbfgs).
 *
 **/
#pragma once
#include "lbfgs.h"
#include "timer.h"

#include <iostream>
#include <limits>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>
#include <hip/device_functions.h>
#include <fstream>
#include <sstream>


using namespace std;

#define MIN(a,b) ((a) < (b) ? (a) : (b))
#define MAX(a,b) ((a) > (b) ? (a) : (b))

namespace gpu_lbfgs {

	// Variables

	__device__ float fkm1;
	__device__ float fk;
	__device__ float tmp;

	__device__ float alpha[HISTORY_SIZE];
	__device__ float rho  [HISTORY_SIZE];
	__device__ float H0;
	__device__ float step;
	__device__ float tmp2;
	__device__ int status;

	// Small helper kernels for scalar operations in device memory needed during updates.
	// What they're used for is documented by comments in the places they are executed.
	// *** Use with a single thread only! ***

	__global__ void update1   (float *alpha_out, const float *sDotZ, const float *rho, float *minusAlpha_out);       // first  update loop
	__global__ void update2   (float *alphaMinusBeta_out, const float *rho, const float *yDotZ, const float *alpha); // second update loop
	__global__ void update3   (float *rho_out, float *H0_out, const float *yDotS, const float *yDotY);               // after line search
}

// linesearch_gpu.h is no real header, it contains
// part of the implementation and must be included
// after the variables above have been declared.
#include "linesearch_gpu.h" 

lbfgs::lbfgs(cost_function& cf, hipblasHandle_t cublasHandle)
	: m_costFunction(cf)
	, m_maxIter(100)
	, m_maxEvals(std::numeric_limits<size_t>::max())
	, m_gradientEps(1e-4f)
	,m_cublasHandle (cublasHandle)
{

	

}

lbfgs::~lbfgs()
{
	
}

std::string lbfgs::statusToString(lbfgs::status stat)
{
	switch (stat)
	{
		case LBFGS_BELOW_GRADIENT_EPS:
			return "Below gradient epsilon";
		case LBFGS_REACHED_MAX_ITER:
			return "Reached maximum number of iterations";
		case LBFGS_REACHED_MAX_EVALS:
			return "Reached maximum number of function/gradient evaluations";
		case LBFGS_LINE_SEARCH_FAILED:
			return "Line search failed";
		default:
			return "Unknown status";
	}
}

lbfgs::status lbfgs::minimize(float *d_x)
{
	return gpu_lbfgs(d_x);
}

lbfgs::status lbfgs::minimize_with_host_x(float *h_x)
{
	 const size_t NX = m_costFunction.getNumberOfUnknowns();
	 float *d_x;
	 hipMalloc((void**)&d_x, NX * sizeof(float));
	 hipMemcpy(d_x, h_x, NX * sizeof(float), hipMemcpyHostToDevice);

	 status ret = minimize(d_x);

	 hipMemcpy(h_x, d_x, NX * sizeof(float), hipMemcpyDeviceToHost);
	 hipFree(d_x);

	 return ret;
}

lbfgs::status lbfgs::gpu_lbfgs(float *d_x)
{
#ifdef LBFGS_TIMING
	timer timer_total     ("GPU_LBFGS_total"     );
	timer timer_evals     ("GPU_LBFGS_evals"     );
	timer timer_updates   ("GPU_LBFGS_updates"   );
	timer timer_linesearch("GPU_LBFGS_linesearch");

	timer_total.start();
#endif

	using namespace gpu_lbfgs;
	const size_t NX = m_costFunction.getNumberOfUnknowns();

	float *d_fkm1, *d_fk;  // f_{k-1}, f_k, function values at x_{k-1} and x_k
	float *d_gkm1, *d_gk;  // g_{k-1}, g_k, gradients       at x_{k-1} and x_k
	float *d_z;            // z,            search direction
	float *d_H0;           // H_0,          initial inverse Hessian (diagonal, same value for all elements)

	float *d_step;         // step          current step length
	float *d_tmp, *d_tmp2; // tmp, tmp2     temporary storage for intermediate results
	int   *d_status;       // status        return code for communication device -> host

	// Ring buffers for history

	float *d_s;            // s,            history of solution updates
	float *d_y;            // y,            history of gradient updates
	float *d_alpha;        // alpha,        history of alphas (needed for z updates)
	float *d_rho;          // rho,          history of rhos   (needed for z updates)

	// Allocations

	CudaSafeCall( hipMalloc(&d_gk,   NX * sizeof(float)) );
	CudaSafeCall( hipMalloc(&d_gkm1, NX * sizeof(float)) );
	CudaSafeCall( hipMalloc(&d_z,    NX * sizeof(float)) );

	CudaSafeCall( hipMalloc(&d_s,    HISTORY_SIZE * NX * sizeof(float)) );
	CudaSafeCall( hipMalloc(&d_y,    HISTORY_SIZE * NX * sizeof(float)) );

	// Addresses of global symbols

	CudaSafeCall( hipGetSymbolAddress((void**)&d_fkm1,   gpu_lbfgs::fkm1  ) );
	CudaSafeCall( hipGetSymbolAddress((void**)&d_fk,     gpu_lbfgs::fk    ) );
	CudaSafeCall( hipGetSymbolAddress((void**)&d_tmp,    gpu_lbfgs::tmp   ) );
	CudaSafeCall( hipGetSymbolAddress((void**)&d_tmp2,   gpu_lbfgs::tmp2  ) );
	CudaSafeCall( hipGetSymbolAddress((void**)&d_H0,     gpu_lbfgs::H0    ) );
	CudaSafeCall( hipGetSymbolAddress((void**)&d_alpha,  gpu_lbfgs::alpha ) );
	CudaSafeCall( hipGetSymbolAddress((void**)&d_rho,    gpu_lbfgs::rho   ) );
	CudaSafeCall( hipGetSymbolAddress((void**)&d_step,   gpu_lbfgs::step  ) );
	CudaSafeCall( hipGetSymbolAddress((void**)&d_status, gpu_lbfgs::status) );

	// Initialize

#ifdef LBFGS_TIMING
	timer_evals.start();
#endif

	m_costFunction.f_gradf(d_x, d_fk, d_gk);

	CudaCheckError();
	hipDeviceSynchronize();

#ifdef LBFGS_TIMING
	timer_evals.stop();
#endif

	size_t evals = 1;

	status stat = LBFGS_REACHED_MAX_ITER;

#ifdef LBFGS_VERBOSE
	std::cout << "lbfgs::gpu_lbfgs()" << std::endl;
#endif

	// H0 = 1.0f;
	const float one = 1.0f;
	CudaSafeCall( hipMemcpy(d_H0, &one, sizeof(float), hipMemcpyHostToDevice) );

	size_t it;

	for (it = 0; it < m_maxIter; ++it)
	{
#ifdef LBFGS_VERBOSE
		float  h_y;
		CudaSafeCall( hipMemcpy(&h_y, d_fk, sizeof(float), hipMemcpyDeviceToHost) );

		float gknorm2;
		dispatch_dot(NX, &gknorm2, d_gk, d_gk, false);

		printf("f(x) = % 12e, ||grad||_2 = % 12e\n", h_y, std::sqrt(gknorm2));
#endif

		// Check for convergence
		// ---------------------

		float gkNormSquared;
		float xkNormSquared;

		dispatch_dot(NX, &xkNormSquared, d_x,  d_x,  false);
		dispatch_dot(NX, &gkNormSquared, d_gk, d_gk, false);

		if (gkNormSquared < (m_gradientEps * m_gradientEps) * max(xkNormSquared, 1.0f))
		{
			stat = LBFGS_BELOW_GRADIENT_EPS;
			break;
		}

		// Find search direction
		// ---------------------

#ifdef LBFGS_TIMING
		timer_updates.start();
#endif

		const float minusOne = -1.0f;
		dispatch_scale(NX, d_z, d_gk, &minusOne, false); // z = -gk

		const size_t MAX_IDX = MIN(it, HISTORY_SIZE);

		for (size_t i = 1; i <= MAX_IDX; ++i)
		{
			size_t idx = index(it - i);

			dispatch_dot(NX, d_tmp, d_s + idx * NX, d_z); // tmp = sDotZ

			// alpha = tmp * rho
			// tmp = -alpha
			update1<<<1, 1>>>(d_alpha + idx, d_tmp, d_rho + idx, d_tmp);

			CudaCheckError();
			hipDeviceSynchronize();

			// z += tmp * y
			dispatch_axpy(NX, d_z, d_z, d_y + idx * NX, d_tmp);
		}

		dispatch_scale(NX, d_z, d_z, d_H0); // z = H0 * z

		for (size_t i = MAX_IDX; i > 0; --i)
		{
			size_t idx = index(it - i);

			dispatch_dot(NX, d_tmp, d_y + idx * NX, d_z); // tmp = yDotZ

			// beta = rho * tmp
			// tmp = alpha - beta
			update2<<<1, 1>>>(d_tmp, d_rho + idx, d_tmp, d_alpha + idx);

			CudaCheckError();
			hipDeviceSynchronize();

			// z += tmp * s
			dispatch_axpy(NX, d_z, d_z, d_s + idx * NX, d_tmp);
		}

#ifdef LBFGS_TIMING
		timer_updates.stop();
		timer_linesearch.start();
#endif

		CudaSafeCall( hipMemcpy(d_fkm1, d_fk, 1  * sizeof(float), hipMemcpyDeviceToDevice) ); // fkm1 = fk;
		CudaSafeCall( hipMemcpy(d_gkm1, d_gk, NX * sizeof(float), hipMemcpyDeviceToDevice) ); // gkm1 = gk;

		timer *t_evals = NULL, *t_linesearch = NULL;
#ifdef LBFGS_TIMING
		t_evals = &timer_evals;
		t_linesearch = &timer_linesearch;
#endif

		// (line search defined in linesearch_gpu.h)
		if (!gpu_linesearch(d_x, d_z, d_fk, d_gk, evals, d_gkm1, d_fkm1, stat, d_step,
							m_maxEvals, t_evals, t_linesearch, d_tmp, d_status))
		{
			break;
		}

#ifdef LBFGS_TIMING
		timer_linesearch.stop();
		timer_updates.start();
#endif

		// Update s, y, rho and H_0
		// ------------------------

		// s   = x_k - x_{k-1} = step * z
		// y   = g_k - g_{k-1}
		// rho = 1 / (y^T s)
		// H_0 = (y^T s) / (y^T y)

		float *d_curS = d_s + index(it) * NX;
		float *d_curY = d_y + index(it) * NX;

		dispatch_scale(NX, d_curS, d_z,  d_step);                   // s = step * z
		dispatch_axpy (NX, d_curY, d_gk, d_gkm1, &minusOne, false); // y = gk - gkm1

		dispatch_dot(NX, d_tmp,  d_curY, d_curS); // tmp  = yDotS
		dispatch_dot(NX, d_tmp2, d_curY, d_curY); // tmp2 = yDotY

		// rho = 1 / tmp
		// if (tmp2 > 1e-5)
		//   H0 = tmp / tmp2
		update3<<<1, 1>>>(d_rho + index(it), d_H0, d_tmp, d_tmp2);

		CudaCheckError();
		hipDeviceSynchronize();

#ifdef LBFGS_TIMING
		timer_updates.stop();
#endif
	}

	// Deallocations

	CudaSafeCall( hipFree(d_gk)   );
	CudaSafeCall( hipFree(d_gkm1) );
	CudaSafeCall( hipFree(d_z)    );

	CudaSafeCall( hipFree(d_s)    );
	CudaSafeCall( hipFree(d_y)    );

#ifdef LBFGS_TIMING
	timer_total.stop();

	timer_total.saveMeasurement();
	timer_evals.saveMeasurement();
	timer_updates.saveMeasurement();
	timer_linesearch.saveMeasurement();
#endif

#ifdef LBFGS_VERBOSE
	std::cout << "Number of iterations: " << it << std::endl;
	std::cout << "Number of function/gradient evaluations: " << evals << std::endl;
	std::cout << "Reason for termination: " << statusToString(stat) << std::endl;
#endif

	return stat;
}

// Vector operations
// -----------------

void lbfgs::dispatch_axpy(const size_t n, float *d_dst, const float *d_y, const float *d_x, const float *a, bool aDevicePointer) const
{
	const hipblasPointerMode_t mode = aDevicePointer ? HIPBLAS_POINTER_MODE_DEVICE
													: HIPBLAS_POINTER_MODE_HOST;

	CublasSafeCall( hipblasSetPointerMode(m_cublasHandle, mode) );

	if (d_dst != d_y)
		CudaSafeCall( hipMemcpy(d_dst, d_y, n * sizeof(float), hipMemcpyDeviceToDevice) );

	CublasSafeCall( hipblasSaxpy(m_cublasHandle, int(n), a, d_x, 1, d_dst, 1) );

	}

void lbfgs::dispatch_scale(const size_t n, float *d_dst, const float *d_x, const float *a, bool aDevicePointer) const
{
	const hipblasPointerMode_t mode = aDevicePointer ? HIPBLAS_POINTER_MODE_DEVICE
													: HIPBLAS_POINTER_MODE_HOST;

	CublasSafeCall( hipblasSetPointerMode(m_cublasHandle, mode) );

	if (d_dst != d_x)
		CudaSafeCall( hipMemcpy(d_dst, d_x, n * sizeof(float), hipMemcpyDeviceToDevice) );

	CublasSafeCall( hipblasSscal(m_cublasHandle, int(n), a, d_dst, 1) );
}


void lbfgs::dispatch_dot(const size_t n, float *dst, const float *d_x, const float *d_y, bool dstDevicePointer) const
{
	const hipblasPointerMode_t mode = dstDevicePointer ? HIPBLAS_POINTER_MODE_DEVICE
													  : HIPBLAS_POINTER_MODE_HOST;

	CublasSafeCall( hipblasSetPointerMode(m_cublasHandle, mode) );

	CublasSafeCall( hipblasSdot(m_cublasHandle, int(n), d_x, 1, d_y, 1, dst) );
}

// -----------------

// Device / kernel functions
// -------------------------

namespace gpu_lbfgs
{
	__global__ void update1(float *alpha_out, const float *sDotZ, const float *rho, float *minusAlpha_out)
	{
		*alpha_out      = *sDotZ * *rho;
		*minusAlpha_out = -*alpha_out;
	}

	__global__ void update2(float *alphaMinusBeta_out, const float *rho, const float *yDotZ, const float *alpha)
	{
		const float beta = *rho * *yDotZ;
		*alphaMinusBeta_out = *alpha - beta;
	}

	__global__ void update3(float *rho_out, float *H0_out, const float *yDotS, const float *yDotY)
	{
		*rho_out = 1.0f / *yDotS;

		if (*yDotY > 1e-5)
			*H0_out = *yDotS / *yDotY;
	}
}

// ------------------

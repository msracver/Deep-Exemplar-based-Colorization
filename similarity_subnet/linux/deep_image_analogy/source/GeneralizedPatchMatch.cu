#include "hip/hip_runtime.h"
﻿

#include "GeneralizedPatchMatch.cuh"
#include "hiprand/hiprand_kernel.h"

__host__ __device__ int clamp(int x, int x_max, int x_min) {//assume x_max >= x_min
	if (x > x_max)
	{
		return x_max;
	}
	else if (x < x_min)
	{
		return x_min;
	}
	else
	{
		return x;
	}
}

__host__ __device__ unsigned int XY_TO_INT(int x, int y) {//r represent the number of 10 degree, x,y - 11 bits, max = 2047, r - max = 36, 6 bits
	return (((y) << 11) | (x));
}
__host__ __device__ int INT_TO_X(unsigned int v) {
	return (v)&((1 << 11) - 1);
}
__host__ __device__ int INT_TO_Y(unsigned int v) {
	return (v >> 11)&((1 << 11) - 1);
}

__host__ __device__ int cuMax(int a, int b) {
	if (a > b) {
		return a;
	}
	else {
		return b;
	}
}
__host__ __device__ int cuMin(int a, int b) {
	if (a < b) {
		return a;
	}
	else {
		return b;
	}
}

__device__ float MycuRand(hiprandState &state) {//random number in cuda, between 0 and 1
	
	 return hiprand_uniform(&state);

}
__device__ void InitcuRand(hiprandState &state) {//random number in cuda, between 0 and 1
	
	int i = threadIdx.x + blockIdx.x * blockDim.x;

    hiprand_init(i, 0, 0, &state);

}

__host__ Mat reconstruct_avg(Mat a, Mat b, unsigned int * ann, int patch_w) {

	Mat c;
	a.copyTo(c);
	for (int ay = 0; ay < a.rows; ay++) {
		for (int ax = 0; ax < a.cols; ax++)
		{
		
			float point_num = 0, *dist_tmp;
			
			dist_tmp = new float[3];

			for (int dc = 0; dc < 3; dc++)
			{
				dist_tmp[dc] = 0;
			}

			for (int dx = -patch_w / 2; dx <= patch_w / 2; dx++) {
				for (int dy = -patch_w / 2; dy <=patch_w / 2; dy++)
				{

					if ((ax + dx) < a.cols && (ax + dx) >= 0 && (ay + dy) < a.rows && (ay + dy) >= 0)
					{

						unsigned int vp = ann[(ay + dy)*a.cols + ax + dx];
						int xp = INT_TO_X(vp), yp = INT_TO_Y(vp);

						if ((xp - dx) < b.cols && (xp - dx) >= 0 && (yp - dy) < b.rows && (yp - dy) >= 0)//a patch that contain this pixel
						{

							for (int dc = 0; dc < 3; dc++)
							{
								dist_tmp[dc] += b.at<Vec3b>(yp - dy, xp - dx).val[dc];
							}

							point_num++;
						}
					}

				}

			}

			for (int dc = 0; dc < 3; dc++)
			{
				c.at<Vec3b>(ay, ax).val[dc] = dist_tmp[dc]/point_num;
			}
			
			delete[] dist_tmp;
		}
	}
	return c;
}

__host__ Mat reconstruct_dflow(Mat a, Mat b, unsigned int * ann, int patch_w) {
	Mat flow;
	a.copyTo(flow);
	for (int ay = 0; ay < a.rows; ay++) {
		for (int ax = 0; ax < a.cols; ax++)
		{
			unsigned int v = ann[ay*a.cols + ax];
			int xbest = INT_TO_X(v);
			int ybest = INT_TO_Y(v);
			flow.at<Vec3b>(ay, ax).val[0] = (uchar)(255 * ((float)(ax - xbest + b.cols - 1) / (2 * b.cols)));
			flow.at<Vec3b>(ay, ax).val[2] = (uchar)(0);
			flow.at<Vec3b>(ay, ax).val[1] = (uchar)(255 * ((float)(ay - ybest + b.rows - 1) / (2 * b.rows)));
		}
	}
	return flow;
}

__host__ __device__ float dist_compute(float * a, float * b, float * a1, float * b1, int channels, int a_rows, int a_cols, int b_rows, int b_cols, int ax, int ay, int bx, int by, int patch_w, float cutoff = INT_MAX) {//this is the average number of all matched pixel
																																																		  //suppose patch_w is an odd number
	float pixel_sum = 0, pixel_no = 0, pixel_dist = 0;//number of pixels realy counted
	float pixel_sum1 = 0;
	int a_slice = a_rows*a_cols, b_slice = b_rows*b_cols;
	int a_pitch = a_cols, b_pitch = b_cols;
	float dp_tmp;

	for (int dy = -patch_w / 2; dy <= patch_w / 2; dy++) {
		for (int dx = -patch_w / 2; dx <= patch_w / 2; dx++) {

			if (
				(ay + dy) < a_rows && (ay + dy) >= 0 && (ax + dx) < a_cols && (ax + dx) >= 0
				&&
				(by + dy) < b_rows && (by + dy) >= 0 && (bx + dx) < b_cols && (bx + dx) >= 0
				)//the pixel in a should exist and pixel in b should exist
			{
				if (channels == 3)
				{
					for (int dc = 0; dc < channels; dc++)
					{
						dp_tmp = a[dc * a_slice + (ay + dy) * a_pitch + (ax + dx)] - b[dc * b_slice + (by + dy) * b_pitch + (bx + dx)];
						pixel_sum += dp_tmp * dp_tmp;
						//dp_tmp = a1[dc * a_slice + (ay + dy) * a_pitch + (ax + dx)] - b1[dc * b_slice + (by + dy) * b_pitch + (bx + dx)];
						//pixel_sum1 += dp_tmp * dp_tmp;
					}
				}
				else
				{
					for (int dc = 0; dc < channels; dc++)
					{
						dp_tmp = a[dc * a_slice + (ay + dy) * a_pitch + (ax + dx)] * b[dc * b_slice + (by + dy) * b_pitch + (bx + dx)];
						pixel_sum -= dp_tmp;
						//dp_tmp = a1[dc * a_slice + (ay + dy) * a_pitch + (ax + dx)] * b1[dc * b_slice + (by + dy) * b_pitch + (bx + dx)];
						//pixel_sum1 -= dp_tmp;
					}
				}
				

				pixel_no += 1;
			}
		}

	}


	pixel_dist = (pixel_sum + pixel_sum1) / pixel_no;
	if (pixel_dist >= cutoff) { return cutoff; }
	else {
		return pixel_dist;
	}
}

__host__ __device__ float dist(float * a, float * b, float *a1, float *b1, int channels, int a_rows, int a_cols, int b_rows, int b_cols, int ax, int ay, int xp, int yp, int patch_w, float cutoff = INT_MAX) {

	return dist_compute(a, b, a1, b1,  channels, a_rows, a_cols, b_rows, b_cols, ax, ay, xp, yp, patch_w, cutoff);

}

__device__ void improve_guess(float * a, float * b, float *a1, float *b1, int channels, int a_rows, int a_cols, int b_rows, int b_cols, int ax, int ay, int &xbest, int &ybest, float &dbest, int xp, int yp, int patch_w, float rr) {
	float d;
	d = dist(a, b, a1, b1, channels, a_rows, a_cols, b_rows, b_cols, ax, ay, xp, yp, patch_w, dbest);
	if (d + rr < dbest) {
		xbest = xp;
		ybest = yp;
		dbest = d;
	}
}


__global__ void initialAnn_kernel(unsigned int * ann, int * params) {

	//just use 7 of 9 parameters
	int ah = params[1];
	int aw = params[2];


	int ax = blockIdx.x*blockDim.x + threadIdx.x;
	int ay = blockIdx.y*blockDim.y + threadIdx.y;

	if (ax < aw && ay < ah) {
		int bx = ax;
		int by = ay;
		ann[ay*aw + ax] = XY_TO_INT(bx, by);
	}
}

__global__ void upSample_kernel(unsigned int * ann, unsigned int * ann_tmp,int * params, int aw_half,int ah_half) {

	int ax = blockIdx.x*blockDim.x + threadIdx.x;
	int ay = blockIdx.y*blockDim.y + threadIdx.y;

	
	int ah = params[1];
	int aw = params[2];
	int bh = params[3];
	int bw = params[4];
	
	
	float aw_ratio = (float)aw / (float)aw_half;
	float ah_ratio = (float)ah / (float)ah_half;
	int ax_half = (ax+0.5) / aw_ratio;
	int ay_half = (ay+0.5) / ah_ratio;
	ax_half = clamp(ax_half, aw_half - 1, 0);
	ay_half = clamp(ay_half, ah_half - 1, 0);
	

	if (ax < aw&&ay < ah) {

		unsigned int v_half = ann[ay_half*aw_half + ax_half];
		int bx_half = INT_TO_X(v_half);
		int by_half = INT_TO_Y(v_half);

		int bx = ax + (bx_half - ax_half)*aw_ratio + 0.5;
		int by = ay + (by_half - ay_half)*ah_ratio + 0.5;

		bx = clamp(bx, bw-1, 0);
		by = clamp(by, bh-1, 0);

		ann_tmp[ay*aw + ax] = XY_TO_INT(bx, by);
	}

}

__global__ void patchmatch(float * a, float * b, float *a1, float *b1, unsigned int *ann, float *annd, int * params) {

	int ax = blockIdx.x*blockDim.x + threadIdx.x;
	int ay = blockIdx.y*blockDim.y + threadIdx.y;

	//assign params
	int ch = params[0];
	int a_rows = params[1];
	int a_cols = params[2];
	int b_rows = params[3];
	int b_cols = params[4];
	int patch_w = params[5];
	int pm_iters = params[6];
	int rs_max = params[7];


	if (ax < a_cols && ay < a_rows) {
	
		// for random number
		hiprandState state;
		InitcuRand(state);

		unsigned int v, vp;

		int xp, yp, xbest, ybest;

		int xmin, xmax, ymin, ymax;

		float dbest;
		v = ann[ay*a_cols + ax];
		xbest = INT_TO_X(v), ybest = INT_TO_Y(v);			
		annd[ay*a_cols + ax] = dist(a, b, a1, b1,  ch, a_rows, a_cols, b_rows, b_cols, ax, ay, xbest, ybest, patch_w);

		for (int iter = 0; iter < pm_iters; iter++) {

			/* Current (best) guess. */
			v = ann[ay*a_cols + ax];
			xbest = INT_TO_X(v), ybest = INT_TO_Y(v);			
			dbest = annd[ay*a_cols + ax];

			/* In each iteration, improve the NNF, by jumping flooding. */
			for (int jump = 8; jump > 0; jump /= 2) {

				/* Propagation: Improve current guess by trying instead correspondences from left, right, up and downs. */
				if ((ax - jump) < a_cols && (ax - jump) >= 0)//left
				{
					vp = ann[ay*a_cols + ax - jump];//the pixel coordinates in image b

					xp = INT_TO_X(vp) + jump, yp = INT_TO_Y(vp);//the propagated match from vp, the center of the patch, which should be in the image

					if (yp >= 0 && yp < b_rows && xp >= 0 && xp < b_cols)
					{
						//improve guess
						improve_guess(a, b, a1, b1, ch, a_rows, a_cols, b_rows, b_cols, ax, ay, xbest, ybest, dbest, xp, yp, patch_w, 0);
						ann[ay*a_cols + ax] = XY_TO_INT(xbest, ybest);
						annd[ay*a_cols + ax] = dbest;
					}
				}
				
				if ((ax + jump) < a_cols)//right
				{
					vp = ann[ay*a_cols + ax + jump];//the pixel coordinates in image b

					xp = INT_TO_X(vp) - jump, yp = INT_TO_Y(vp);

					if (yp >= 0 && yp < b_rows && xp >= 0 && xp < b_cols)
					{
						//improve guess
						improve_guess(a, b, a1, b1, ch, a_rows, a_cols, b_rows, b_cols, ax, ay, xbest, ybest, dbest, xp, yp, patch_w, 0);
						ann[ay*a_cols + ax] = XY_TO_INT(xbest, ybest);
						annd[ay*a_cols + ax] = dbest;
					}
				}

				if ((ay - jump) < a_rows && (ay - jump) >= 0)//up
				{
					vp = ann[(ay - jump)*a_cols + ax];//the pixel coordinates in image b
					xp = INT_TO_X(vp), yp = INT_TO_Y(vp) + jump;

					if (yp >= 0 && yp < b_rows && xp >= 0 && xp < b_cols)
					{

						//improve guess
						improve_guess(a, b, a1, b1, ch, a_rows, a_cols, b_rows, b_cols, ax, ay, xbest, ybest, dbest, xp, yp, patch_w, 0);
						ann[ay*a_cols + ax] = XY_TO_INT(xbest, ybest);
						annd[ay*a_cols + ax] = dbest;
					}
				}

				if ((ay + jump) < a_rows)//down
				{
					vp = ann[(ay + jump)*a_cols + ax];//the pixel coordinates in image b	
					xp = INT_TO_X(vp), yp = INT_TO_Y(vp) - jump;

					if (yp >= 0 && yp < b_rows && xp >= 0 && xp < b_cols)
					{
						//improve guess
						improve_guess(a, b, a1, b1,  ch, a_rows, a_cols, b_rows, b_cols, ax, ay, xbest, ybest, dbest, xp, yp, patch_w, 0);
						ann[ay*a_cols + ax] = XY_TO_INT(xbest, ybest);
						annd[ay*a_cols + ax] = dbest;
					}
				}

			}

			/* Random search: Improve current guess by searching in boxes of exponentially decreasing size around the current best guess. */
			int rs_start = rs_max;
			if (rs_start > cuMax(b_cols, b_rows)) {
				rs_start = cuMax(b_cols, b_rows);
			}
			for (int mag = rs_start; mag >= 1; mag /= 2) {
				/* Sampling window */
				xmin = cuMax(xbest - mag, 0), xmax = cuMin(xbest + mag + 1, b_cols);
				ymin = cuMax(ybest - mag, 0), ymax = cuMin(ybest + mag + 1, b_rows);
				xp = xmin + (int)(MycuRand(state)*(xmax - xmin)) % (xmax - xmin);
				yp = ymin + (int)(MycuRand(state)*(ymax - ymin)) % (ymax - ymin);

				//improve guess
				improve_guess(a, b, a1, b1,  ch, a_rows, a_cols, b_rows, b_cols, ax, ay, xbest, ybest, dbest, xp, yp, patch_w, FLT_MIN);

			}

			ann[ay*a_cols + ax] = XY_TO_INT(xbest, ybest);
			annd[ay*a_cols + ax] = dbest;
			__syncthreads();
		}
	}
}

__global__ void initialAnn_kernel(unsigned int * ann, int * params, int w, int h) {

	//just use 7 of 9 parameters
	int ah = params[1];
	int aw = params[2];


	int ax = blockIdx.x*blockDim.x + threadIdx.x;
	int ay = blockIdx.y*blockDim.y + threadIdx.y;

	if (ax < aw && ay < ah) {
		int bx = min(int(ax / (float)aw * w), w - 1);
		int by = min(int(ay / (float)ah * h), h - 1);
		ann[ay*aw + ax] = XY_TO_INT(bx, by);
	}
}


__global__ void blend(float *cmap, float* oldd, float* newd, float weight,int * params)
{
	int ax = blockIdx.x*blockDim.x + threadIdx.x;
	int ay = blockIdx.y*blockDim.y + threadIdx.y;

	int ch = params[0];
	int ah = params[1];
	int aw = params[2];
	
	int slice_a = ah * aw;
	int pitch_a = aw;
	
	float thre = 0.05;
	
	if (ax < aw&& ay < ah)
	{
		float fa = cmap[ay*pitch_a + ax];
		if (fa < thre)
			fa = 0.0f;
		else fa = weight;
		for (int i = 0; i < ch; i++)
		{
			
			newd[i*slice_a + ay*pitch_a + ax] = oldd[i*slice_a + ay*pitch_a + ax]* fa + newd[i*slice_a + ay*pitch_a + ax] * (1.0-fa);
		}
	}
}

// ********** VOTE ***********

__global__ void avg_vote(unsigned int * ann, float * pb, float * pc, int * params) {

	int ax = blockIdx.x*blockDim.x + threadIdx.x;
	int ay = blockIdx.y*blockDim.y + threadIdx.y;

	int ch = params[0];
	int ah = params[1];
	int aw = params[2];
	int bh = params[3];
	int bw = params[4];
	int patch_w = params[5];

	int slice_a = ah * aw;
	int pitch_a = aw;
	int slice_b = bh * bw;
	int pitch_b = bw;

	int count = 0;

	if (ax < aw&&ay < ah)
	{

		//set zero for all the channels at (ax,ay)
		for (int i = 0; i < ch; i++)
		{
			pc[i*slice_a + ay*pitch_a + ax] = 0;

		}

		//count the sum of all the possible value of (ax,ay)
		for (int dx = -patch_w / 2; dx <= patch_w / 2; dx++) {
			for (int dy = -patch_w / 2; dy <= patch_w / 2; dy++)
			{

				if ((ax + dx) < aw && (ax + dx) >= 0 && (ay + dy) < ah && (ay + dy) >= 0)
				{
					unsigned int vp = ann[(ay + dy)*aw + ax + dx];
					
					int xp = INT_TO_X(vp);
					int yp = INT_TO_Y(vp);

					if ((xp - dx) < bw && (xp - dx) >= 0 && (yp - dy) < bh && (yp - dy) >= 0)
					{
						count++;
						for (int dc = 0; dc < ch; dc++)
						{
							pc[dc*slice_a + ay*pitch_a + ax] += pb[dc*slice_b + (yp - dy)*pitch_b + xp - dx];
						}
					}
				}

			}
		}

		//count average value
		for (int i = 0; i < ch; i++)
		{
			pc[i*slice_a + ay*pitch_a + ax] /= count;
		}

	}
}

#include "hip/hip_runtime.h"
#include "caffe/util/math_functions.hpp"
#include "caffe/common.hpp"
#include "GeneralizedPatchMatch.cuh"
#include "DeepAnalogy.cuh"
#include "WLS.h"
#include "Deconv.h"

struct Parameters
{

	std::vector<std::string> layers; //which layers  used as content

	int patch_size0;
	int iter;

};


__host__ void norm(float* &dst, float* src, float* smooth, Dim dim){

	int count = dim.channel*dim.height*dim.width;
	float* x = src;
	float* x2;
	hipMalloc(&x2, count*sizeof(float));
	caffe_gpu_mul(count, x, x, x2);

	//caculate dis
	float*sum;
	float* ones;
	hipMalloc(&sum, dim.height*dim.width*sizeof(float));
	hipMalloc(&ones, dim.channel*sizeof(float));
	caffe_gpu_set(dim.channel, 1.0f, ones);
	caffe_gpu_gemv(CblasTrans, dim.channel, dim.height*dim.width, 1.0f, x2, ones, 0.0f, sum);

	float *dis;
	hipMalloc(&dis, dim.height*dim.width*sizeof(float));
	caffe_gpu_powx(dim.height*dim.width, sum, 0.5f, dis);

	if (smooth != NULL)
	{
		hipMemcpy(smooth, sum, dim.height*dim.width*sizeof(float), hipMemcpyDeviceToDevice);
		int index;
		float minv, maxv;
		hipblasIsamin(Caffe::cublas_handle(), dim.height*dim.width, sum, 1, &index);
		hipMemcpy(&minv, sum + index - 1, sizeof(float), hipMemcpyDeviceToHost);
		hipblasIsamax(Caffe::cublas_handle(), dim.height*dim.width, sum, 1, &index);
		hipMemcpy(&maxv, sum + index - 1, sizeof(float), hipMemcpyDeviceToHost);

		caffe_gpu_add_scalar(dim.height*dim.width, -minv, smooth);
		caffe_gpu_scal(dim.height*dim.width, 1.0f / (maxv - minv), smooth);
	}


	//norm
	caffe_gpu_gemm(CblasNoTrans, CblasNoTrans, dim.channel, dim.width*dim.height, 1, 1.0f, ones, dis, 0.0f, x2);
	caffe_gpu_div(count, src, x2, dst);

	hipFree(x2);
	hipFree(ones);
	hipFree(dis);
	hipFree(sum);
}

DeepAnalogy::DeepAnalogy(){
	resizeRatio = 1;
	weightLevel = 3;
	photoTransfer = false;
	file_A = "";
	file_BP = "";
	path_output = "";
	path_model = "";


}

DeepAnalogy::~DeepAnalogy(){

}

void DeepAnalogy::SetRatio(float ratio){
	resizeRatio = ratio;
}
void DeepAnalogy::SetBlendWeight(int level){
	weightLevel = level;
}
void DeepAnalogy::UsePhotoTransfer(bool flag){
	photoTransfer = flag;
}
void DeepAnalogy::SetModel(string path){
	path_model =path;
}
void DeepAnalogy::SetA(string f_a){
	file_A = f_a;
}
void DeepAnalogy::SetBPrime(string f_bp){
	file_BP = f_bp;
}
void DeepAnalogy::SetOutputDir(string f_o){
	path_output = f_o;
}
void DeepAnalogy::SetGPU(int no){
	hipSetDevice(no);
}
void DeepAnalogy::LoadInputs(){
	float ratio;
	Mat ori_AL = imread(file_A);
	Mat ori_BPL = imread(file_BP);
	if (ori_AL.empty() || ori_BPL.empty())
	{
		cout << "image cannot read!" << endl;
		waitKey();
		return;
	}

	ori_A_cols = ori_AL.cols;
	ori_A_rows = ori_AL.rows;
	ori_BP_cols = ori_BPL.cols;
	ori_BP_rows = ori_BPL.rows;

	if (ori_AL.rows > 700)
	{
		ratio = 700.f / ori_AL.rows;
		cv::resize(ori_AL, img_AL, Size(), ratio, ratio, INTER_CUBIC);
		ori_AL = img_AL.clone();

	}
	if (ori_AL.cols > 700)
	{
		ratio = 700.f / ori_AL.cols;
		cv::resize(ori_AL, img_AL, Size(), ratio, ratio, INTER_CUBIC);
		ori_AL = img_AL.clone();

	}

	if (ori_AL.rows < 200)
	{
		ratio = 200.f / ori_AL.rows;
		cv::resize(ori_AL, img_AL, Size(), ratio, ratio, INTER_CUBIC);
		ori_AL = img_AL.clone();

	}

	if (ori_AL.cols < 200)
	{
		ratio = 200.f / ori_AL.cols;
		cv::resize(ori_AL, img_AL, Size(), ratio, ratio, INTER_CUBIC);
		ori_AL = img_AL.clone();

	}

	if (ori_BPL.rows > 700)
	{
		ratio = 700.f / ori_BPL.rows;
		cv::resize(ori_BPL, img_BPL, Size(), ratio, ratio, INTER_CUBIC);
		ori_BPL = img_BPL.clone();

	}

	if (ori_BPL.cols > 700)
	{
		ratio = 700.f / ori_BPL.cols;
		cv::resize(ori_BPL, img_BPL, Size(), ratio, ratio, INTER_CUBIC);
		ori_BPL = img_BPL.clone();
	}
	if (ori_BPL.rows < 200)
	{
		ratio = 200.f / ori_BPL.rows;
		cv::resize(ori_BPL, img_BPL, Size(), ratio, ratio, INTER_CUBIC);
		ori_BPL = img_BPL.clone();

	}

	if (ori_BPL.cols < 200)
	{
		ratio = 200.f / ori_BPL.cols;
		cv::resize(ori_BPL, img_BPL, Size(), ratio, ratio, INTER_CUBIC);
		ori_BPL = img_BPL.clone();

	}

	if ((ori_AL.cols*ori_AL.rows) > 350000)
	{
		ratio = sqrt((float)(350000) / (float)(ori_AL.cols*ori_AL.rows));
		cv::resize(ori_AL, img_AL, Size(), ratio, ratio, INTER_CUBIC);
		ori_AL = img_AL.clone();

	}

	if ((ori_BPL.cols*ori_BPL.rows) > 350000)
	{
		ratio = sqrt((float)(350000) / (float)(ori_BPL.cols*ori_BPL.rows));
		cv::resize(ori_BPL, img_BPL, Size(), ratio, ratio, INTER_CUBIC);
		ori_BPL = img_BPL.clone();
	}


	int maxLateral, minLateral;
	maxLateral = max(max(ori_AL.rows, ori_AL.cols), max(ori_BPL.rows, ori_BPL.cols));
	minLateral = min(min(ori_AL.rows, ori_AL.cols), min(ori_BPL.rows, ori_BPL.cols));

	if (maxLateral > 700 || minLateral < 200)
	{
		cout << "The sizes of images are not permitted. (One side cannot be larger than 700 or smaller than 200 and the area should not be larger than 350000)" << endl;
		waitKey();
		return;
	}


	cur_A_cols = ori_AL.cols;
	cur_A_rows = ori_AL.rows;
	cur_BP_cols = ori_BPL.cols;
	cur_BP_rows = ori_BPL.rows;

	if (ori_A_cols != ori_AL.cols)
	{
		cout << "The input image A has been resized to " << cur_A_cols << " x " << cur_A_rows << ".\n";
	}

	if (ori_BP_cols != ori_BPL.cols)
	{
		cout << "The input image B prime has been resized to " << cur_BP_cols << " x " << cur_BP_rows << ".\n";
	}


	cv::resize(ori_AL, img_AL, Size(), (float)cur_A_cols / ori_AL.cols, (float)cur_A_rows / ori_AL.rows, INTER_CUBIC);
	cv::resize(ori_BPL, img_BPL, Size(), (float)cur_BP_cols / ori_BPL.cols, (float)cur_BP_rows / ori_BPL.rows, INTER_CUBIC);


}

void DeepAnalogy::ComputeAnn(Classifier& classifier_A, Classifier& classifier_B) {

	if (img_BPL.empty()||img_AL.empty())
	{
		waitKey();
		return;
	}

	const int param_size = 8;


	int ann_size_AB, ann_size_BA;//should be assigned later
	int *params_host, *params_device_AB, *params_device_BA;
	unsigned int *ann_device_AB, *ann_host_AB, *ann_device_BA, *ann_host_BA;
	float *annd_device_AB, *annd_host_AB, *annd_device_BA, *annd_host_BA;

	char fname[256];

	//set parameters
	Parameters params;
	params.layers.push_back("conv5_1");
	params.layers.push_back("conv4_1");
	params.layers.push_back("conv3_1");
	params.layers.push_back("conv2_1");
	params.layers.push_back("conv1_1");
	params.layers.push_back("data");

	std::vector<float> weight;
	weight.push_back(1.0);
	switch (weightLevel)
	{
	case 1:
		weight.push_back(0.7);
		weight.push_back(0.6);
		weight.push_back(0.5);
		weight.push_back(0.0);
		break;
	case 2:
		weight.push_back(0.8);
		weight.push_back(0.7);
		weight.push_back(0.6);
		weight.push_back(0.1);
		break;

	case 3:
		weight.push_back(0.9);
		weight.push_back(0.8);
		weight.push_back(0.7);
		weight.push_back(0.2);
		break;

	default:
		weight.push_back(0.9);
		weight.push_back(0.8);
		weight.push_back(0.7);
		weight.push_back(0.2);
		break;
	}

	weight.push_back(0.0);

	std::vector<int> sizes;
	sizes.push_back(3);
	sizes.push_back(3);
	sizes.push_back(3);
	sizes.push_back(5);
	sizes.push_back(5);
	sizes.push_back(3);

	params.iter = 10;

	//scale and enhance
    float ratio = resizeRatio;
	Mat img_BP, img_A;
	cv::resize(img_AL, img_A, Size(), ratio, ratio, INTER_CUBIC);
	cv::resize(img_BPL, img_BP, Size(), ratio, ratio, INTER_CUBIC);

	std::vector<int> range;
	if (img_A.cols > img_A.rows)
	{
		range.push_back(img_A.cols / 16);

	}
	else
	{
		range.push_back(img_A.rows / 16);

	}
	range.push_back(6);
	range.push_back(6);
	range.push_back(4);
	range.push_back(4);
	range.push_back(2);

	//load caffe
	//::google::InitGoogleLogging("deepanalogy");
	//string model_file = "vgg19/VGG_ILSVRC_19_layers_deploy.prototxt";
	//string trained_file = "vgg19/VGG_ILSVRC_19_layers.caffemodel";

	//Classifier classifier_A(path_model + model_file, path_model + trained_file);
	//Classifier classifier_B(path_model + model_file, path_model + trained_file);


	std::vector<float *> data_A, data_AP;
	data_A.resize(params.layers.size());
	data_AP.resize(params.layers.size());
	std::vector<Dim> data_A_size;
	data_A_size.resize(params.layers.size());
	classifier_A.Predict(img_A, params.layers, data_AP, data_A, data_A_size);

	std::vector<float *> data_B, data_BP;
	data_B.resize(params.layers.size());
	data_BP.resize(params.layers.size());
	std::vector<Dim> data_B_size;
	data_B_size.resize(params.layers.size());
	classifier_B.Predict(img_BP, params.layers, data_B, data_BP, data_B_size);

	clock_t start, finish;
	double duration;
	start = clock();

	ann_size_AB = img_AL.cols*img_AL.rows;
	ann_size_BA = img_BPL.cols*img_BPL.rows;
	params_host = (int *)malloc(param_size * sizeof(int));
	ann_host_AB = (unsigned int *)malloc(ann_size_AB * sizeof(unsigned int));
	annd_host_AB = (float *)malloc(ann_size_AB * sizeof(float));
	ann_host_BA = (unsigned int *)malloc(ann_size_BA * sizeof(unsigned int));
	annd_host_BA = (float *)malloc(ann_size_BA * sizeof(float));

	hipMalloc(&params_device_AB, param_size * sizeof(int));
	hipMalloc(&params_device_BA, param_size * sizeof(int));
	hipMalloc(&ann_device_AB, ann_size_AB * sizeof(unsigned int));
	hipMalloc(&annd_device_AB, ann_size_AB * sizeof(float));
	hipMalloc(&ann_device_BA, ann_size_BA * sizeof(unsigned int));
	hipMalloc(&annd_device_BA, ann_size_BA * sizeof(float));

	int numlayer = params.layers.size();


	//feature match
	for (int curr_layer = 0; curr_layer < numlayer - 2; curr_layer++)//from 32 to 512
	{

		//set parameters
		params_host[0] = data_A_size[curr_layer].channel;//channels
		params_host[1] = data_A_size[curr_layer].height;
		params_host[2] = data_A_size[curr_layer].width;
		params_host[3] = data_B_size[curr_layer].height;
		params_host[4] = data_B_size[curr_layer].width;
		params_host[5] = sizes[curr_layer];
		params_host[6] = params.iter;
		params_host[7] = range[curr_layer];

		//copy to device
		hipMemcpy(params_device_AB, params_host, param_size * sizeof(int), hipMemcpyHostToDevice);

		//set parameters
		params_host[0] = data_B_size[curr_layer].channel;//channels
		params_host[1] = data_B_size[curr_layer].height;
		params_host[2] = data_B_size[curr_layer].width;
		params_host[3] = data_A_size[curr_layer].height;
		params_host[4] = data_A_size[curr_layer].width;

		//copy to device
		hipMemcpy(params_device_BA, params_host, param_size * sizeof(int), hipMemcpyHostToDevice);

		////set device pa, device pb, device ann and device annd
		dim3 blocksPerGridAB(data_A_size[curr_layer].width / 20 + 1, data_A_size[curr_layer].height / 20 + 1, 1);
		dim3 threadsPerBlockAB(20, 20, 1);
		ann_size_AB = data_A_size[curr_layer].width* data_A_size[curr_layer].height;
		dim3 blocksPerGridBA(data_B_size[curr_layer].width / 20 + 1, data_B_size[curr_layer].height / 20 + 1, 1);
		dim3 threadsPerBlockBA(20, 20, 1);
		ann_size_BA = data_B_size[curr_layer].width* data_B_size[curr_layer].height;

		//initialize ann if needed
		if (curr_layer == 0)//initialize, rows and cols both less than 32, just use one block
		{

			initialAnn_kernel << <blocksPerGridAB, threadsPerBlockAB >> >(ann_device_AB, params_device_AB, data_B_size[curr_layer].width, data_B_size[curr_layer].height);
			initialAnn_kernel << <blocksPerGridBA, threadsPerBlockBA >> >(ann_device_BA, params_device_BA, data_A_size[curr_layer].width, data_A_size[curr_layer].height);

		}
		else {//upsampling, notice this block's dimension is twice the ann at this point
			unsigned int * ann_tmp;

			hipMalloc(&ann_tmp, ann_size_AB * sizeof(unsigned int));
			upSample_kernel << <blocksPerGridAB, threadsPerBlockAB >> >(ann_device_AB, ann_tmp, params_device_AB,
				data_A_size[curr_layer - 1].width, data_A_size[curr_layer - 1].height);//get new ann_device
			hipMemcpy(ann_device_AB, ann_tmp, ann_size_AB * sizeof(unsigned int), hipMemcpyDeviceToDevice);
			hipFree(ann_tmp);

			hipMalloc(&ann_tmp, ann_size_BA * sizeof(unsigned int));
			upSample_kernel << <blocksPerGridBA, threadsPerBlockBA >> >(ann_device_BA, ann_tmp, params_device_BA,
				data_B_size[curr_layer - 1].width, data_B_size[curr_layer - 1].height);//get new ann_device
			hipMemcpy(ann_device_BA, ann_tmp, ann_size_BA * sizeof(unsigned int), hipMemcpyDeviceToDevice);
			hipFree(ann_tmp);

		}

		//normarlize two data
		float *Ndata_A, *Ndata_AP, *Ndata_B, *Ndata_BP;
		float *response_A, *response_BP;

		hipMalloc(&Ndata_A, data_A_size[curr_layer].channel*data_A_size[curr_layer].width*data_A_size[curr_layer].height*sizeof(float));
		hipMalloc(&Ndata_AP, data_A_size[curr_layer].channel*data_A_size[curr_layer].width*data_A_size[curr_layer].height*sizeof(float));
		hipMalloc(&response_A, data_A_size[curr_layer].width*data_A_size[curr_layer].height*sizeof(float));
		hipMalloc(&Ndata_B, data_B_size[curr_layer].channel*data_B_size[curr_layer].width*data_B_size[curr_layer].height*sizeof(float));
		hipMalloc(&Ndata_BP, data_B_size[curr_layer].channel*data_B_size[curr_layer].width*data_B_size[curr_layer].height*sizeof(float));
		hipMalloc(&response_BP, data_B_size[curr_layer].width*data_B_size[curr_layer].height*sizeof(float));


		norm(Ndata_A, data_A[curr_layer], response_A, data_A_size[curr_layer]);
		norm(Ndata_BP, data_BP[curr_layer], response_BP, data_B_size[curr_layer]);


		Mat temp1, temp2;
		cv::resize(img_AL, temp1, cv::Size(data_A_size[curr_layer].width, data_A_size[curr_layer].height));
		cv::resize(img_BPL, temp2, cv::Size(data_B_size[curr_layer].width, data_B_size[curr_layer].height));

		Mat response1, response2;
		response1 = Mat(temp1.size(), CV_32FC1);
		response2 = Mat(temp2.size(), CV_32FC1);

		hipMemcpy(response1.data, response_A, data_A_size[curr_layer].width*data_A_size[curr_layer].height*sizeof(float), hipMemcpyDeviceToHost);
		hipMemcpy(response2.data, response_BP, data_B_size[curr_layer].width*data_B_size[curr_layer].height*sizeof(float), hipMemcpyDeviceToHost);

		Mat response_byte1, response_byte2;
		response1.convertTo(response_byte1, CV_8UC1, 255);
		response2.convertTo(response_byte2, CV_8UC1, 255);

		blend << <blocksPerGridAB, threadsPerBlockAB >> >(response_A, data_A[curr_layer], data_AP[curr_layer], weight[curr_layer], params_device_AB);
		blend << <blocksPerGridBA, threadsPerBlockBA >> >(response_BP, data_BP[curr_layer], data_B[curr_layer], weight[curr_layer], params_device_BA);

		norm(Ndata_AP, data_AP[curr_layer], NULL, data_A_size[curr_layer]);
		norm(Ndata_B, data_B[curr_layer], NULL, data_B_size[curr_layer]);

		//patchmatch
		cout << "Finding nearest neighbor field using PatchMatch Algorithm at layer:" << params.layers[curr_layer] << ".\n";
		patchmatch << <blocksPerGridAB, threadsPerBlockAB >> >(Ndata_AP, Ndata_BP, Ndata_A, Ndata_B, ann_device_AB, annd_device_AB, params_device_AB);
		patchmatch << <blocksPerGridBA, threadsPerBlockBA >> >(Ndata_B, Ndata_A, Ndata_BP, Ndata_AP, ann_device_BA, annd_device_BA, params_device_BA);

		hipFree(Ndata_A);
		hipFree(Ndata_AP);
		hipFree(Ndata_B);
		hipFree(Ndata_BP);
		hipFree(response_A);
		hipFree(response_BP);

		//deconv
		if (curr_layer < numlayer - 2)
		{
			int next_layer = curr_layer + 2;

			//set parameters			
			params_host[0] = data_A_size[curr_layer].channel;//channels
			params_host[1] = data_A_size[curr_layer].height;
			params_host[2] = data_A_size[curr_layer].width;
			params_host[3] = data_B_size[curr_layer].height;
			params_host[4] = data_B_size[curr_layer].width;
			params_host[5] = sizes[curr_layer];
			params_host[6] = params.iter;
			params_host[7] = range[curr_layer];

			//copy to device
			hipMemcpy(params_device_AB, params_host, param_size * sizeof(int), hipMemcpyHostToDevice);

			//set parameters
			params_host[0] = data_B_size[curr_layer].channel;//channels
			params_host[1] = data_B_size[curr_layer].height;
			params_host[2] = data_B_size[curr_layer].width;
			params_host[3] = data_A_size[curr_layer].height;
			params_host[4] = data_A_size[curr_layer].width;

			//copy to device
			hipMemcpy(params_device_BA, params_host, param_size * sizeof(int), hipMemcpyHostToDevice);

			////set device pa, device pb, device ann and device annd
			blocksPerGridAB = dim3(data_A_size[curr_layer].width / 20 + 1, data_A_size[curr_layer].height / 20 + 1, 1);
			threadsPerBlockAB = dim3(20, 20, 1);
			ann_size_AB = data_A_size[curr_layer].width* data_A_size[curr_layer].height;
			blocksPerGridBA = dim3(data_B_size[curr_layer].width / 20 + 1, data_B_size[curr_layer].height / 20 + 1, 1);
			threadsPerBlockBA = dim3(20, 20, 1);
			ann_size_BA = data_B_size[curr_layer].width* data_B_size[curr_layer].height;

			int num1 = data_A_size[curr_layer].channel*data_A_size[curr_layer].width*data_A_size[curr_layer].height;
			int num2 = data_A_size[next_layer].channel*data_A_size[next_layer].width*data_A_size[next_layer].height;

			float *target;
			hipMalloc(&target, num1 * sizeof(float));
			avg_vote << <blocksPerGridAB, threadsPerBlockAB >> >(ann_device_AB, data_BP[curr_layer], target, params_device_AB);
			deconv(&classifier_A, params.layers[curr_layer], target, data_A_size[curr_layer], params.layers[next_layer], data_AP[next_layer], data_A_size[next_layer]);
			hipFree(target);

			num1 = data_B_size[curr_layer].channel*data_B_size[curr_layer].width*data_B_size[curr_layer].height;
			num2 = data_B_size[next_layer].channel*data_B_size[next_layer].width*data_B_size[next_layer].height;
			hipMalloc(&target, num1 * sizeof(float));
			avg_vote << <blocksPerGridBA, threadsPerBlockBA >> >(ann_device_BA, data_A[curr_layer], target, params_device_BA);
			deconv(&classifier_B, params.layers[curr_layer], target, data_B_size[curr_layer], params.layers[next_layer], data_B[next_layer], data_B_size[next_layer]);
			hipFree(target);

		}


	}

	//upsample
	int curr_layer = numlayer - 1;
	{
		//set parameters
		params_host[0] = 3;//channels
		params_host[1] = img_AL.rows;
		params_host[2] = img_AL.cols;
		params_host[3] = img_BPL.rows;
		params_host[4] = img_BPL.cols;
		params_host[5] = sizes[curr_layer];
		params_host[6] = params.iter;
		params_host[7] = range[curr_layer];
		//copy to device
		hipMemcpy(params_device_AB, params_host, param_size * sizeof(int), hipMemcpyHostToDevice);

		//set parameters
		params_host[0] = 3;//channels
		params_host[1] = img_BPL.rows;
		params_host[2] = img_BPL.cols;
		params_host[3] = img_AL.rows;
		params_host[4] = img_AL.cols;
		//copy to device
		hipMemcpy(params_device_BA, params_host, param_size * sizeof(int), hipMemcpyHostToDevice);

		////set device pa, device pb, device ann and device annd
		dim3 blocksPerGridAB(img_AL.cols / 20 + 1, img_AL.rows / 20 + 1, 1);
		dim3 threadsPerBlockAB(20, 20, 1);
		ann_size_AB = img_AL.cols* img_AL.rows;
		dim3 blocksPerGridBA(img_BPL.cols / 20 + 1, img_BPL.rows / 20 + 1, 1);
		dim3 threadsPerBlockBA(20, 20, 1);
		ann_size_BA = img_BPL.rows* img_BPL.cols;


		//updample
		unsigned int * ann_tmp;
		hipMalloc(&ann_tmp, ann_size_AB * sizeof(unsigned int));
		upSample_kernel << <blocksPerGridAB, threadsPerBlockAB >> >(ann_device_AB, ann_tmp, params_device_AB,
			data_A_size[curr_layer - 2].width, data_A_size[curr_layer - 2].height);//get new ann_device
		hipMemcpy(ann_device_AB, ann_tmp, ann_size_AB * sizeof(unsigned int), hipMemcpyDeviceToDevice);
		hipFree(ann_tmp);

		hipMalloc(&ann_tmp, ann_size_BA * sizeof(unsigned int));
		upSample_kernel << <blocksPerGridBA, threadsPerBlockBA >> >(ann_device_BA, ann_tmp, params_device_BA,
			data_B_size[curr_layer - 2].width, data_B_size[curr_layer - 2].height);//get new ann_device
		hipMemcpy(ann_device_BA, ann_tmp, ann_size_BA * sizeof(unsigned int), hipMemcpyDeviceToDevice);
		hipFree(ann_tmp);

		hipMemcpy(ann_host_AB, ann_device_AB, ann_size_AB * sizeof(unsigned int), hipMemcpyDeviceToHost);
		hipMemcpy(ann_host_BA, ann_device_BA, ann_size_BA * sizeof(unsigned int), hipMemcpyDeviceToHost);

		if (0)
		{
			//save result
			Mat flow, result_AB, result_BA, err, out, normal;

			flow = reconstruct_dflow(img_AL, img_BPL, ann_host_AB, sizes[curr_layer]);
			result_AB = reconstruct_avg(img_AL, img_BPL, ann_host_AB, sizes[curr_layer]);

			cv::resize(result_AB, out, Size(), (float)ori_A_cols / cur_A_cols, (float)ori_A_rows / cur_A_rows, INTER_CUBIC);
			sprintf(fname, "%s\\resultAB.png", path_output.c_str());
			imwrite(fname, out);

			flow = reconstruct_dflow(img_BPL, img_AL, ann_host_BA, sizes[curr_layer]);
			result_BA = reconstruct_avg(img_BPL, img_AL, ann_host_BA, sizes[curr_layer]);

			cv::resize(result_BA, out, Size(), (float)ori_BP_cols / cur_BP_cols, (float)ori_BP_rows / cur_BP_rows, INTER_CUBIC);
			sprintf(fname, "%s\\resultBA.png", path_output.c_str());
			imwrite(fname, out);
		}
	}

	cout << "Saving flow result." << "\n";

	//save ann
	{
		int pos0 = file_A.find_last_of("\\/");
		int pos1 = file_A.find_last_of(".");
		string name_A = file_A.substr(pos0 + 1, pos1 - pos0 - 1);
		pos0 = file_BP.find_last_of("\\/");
		pos1 = file_BP.find_last_of(".");
		string name_B = file_BP.substr(pos0 + 1, pos1 - pos0 - 1);

		ofstream output1;
		char fname[256];
		sprintf(fname, "%s_%s.txt", name_A.c_str(), name_B.c_str());
		output1.open(path_output + fname);
		for (int y = 0; y < img_AL.rows; y++)
		for (int x = 0; x < img_AL.cols; x++)
		{
			unsigned int v = ann_host_AB[y*img_AL.cols + x];
			int xbest = INT_TO_X(v);
			int ybest = INT_TO_Y(v);
			output1 << xbest - x << " " << ybest - y << endl;
		}
		output1.close();

		ofstream output2;
		sprintf(fname, "%s_%s.txt", name_B.c_str(), name_A.c_str());
		output2.open(path_output + fname);
		for (int y = 0; y < img_BPL.rows; y++){
			for (int x = 0; x < img_BPL.cols; x++)
			{
				unsigned int v = ann_host_BA[y*img_BPL.cols + x];
				int xbest = INT_TO_X(v);
				int ybest = INT_TO_Y(v);
				output2 << xbest - x << " " << ybest - y << endl;
			}
		}
		output2.close();
	}

	hipFree(params_device_AB);
	hipFree(ann_device_AB);
	hipFree(annd_device_AB);
	hipFree(params_device_BA);
	hipFree(ann_device_BA);
	hipFree(annd_device_BA);

	free(ann_host_AB);
	free(annd_host_AB);
	free(ann_host_BA);
	free(annd_host_BA);
	free(params_host);

	for (int i = 0; i < numlayer; i++)
	{
		hipFree(data_A[i]);
		hipFree(data_BP[i]);
	}

	finish = clock();
	duration = (double)(finish - start) / CLOCKS_PER_SEC;
	cout << "Finished finding ann. Time : " << duration << endl;

	//google::ShutdownGoogleLogging();
	//classifier_A.DeleteNet();
	//classifier_B.DeleteNet();
}


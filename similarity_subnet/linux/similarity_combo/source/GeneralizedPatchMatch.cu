#include "hip/hip_runtime.h"
﻿// Copyright (c) Microsoft. All rights reserved.

// Licensed under the MIT license. See LICENSE file in the project root for full license information.


#include "GeneralizedPatchMatch.cuh"
#include "hiprand/hiprand_kernel.h"

__host__ __device__ int clamp(int x, int x_max, int x_min) {//assume x_max >= x_min
	if (x > x_max)
	{
		return x_max;
	}
	else if (x < x_min)
	{
		return x_min;
	}
	else
	{
		return x;
	}
}

__host__ __device__ unsigned int XY_TO_INT(int x, int y) {//r represent the number of 10 degree, x,y - 11 bits, max = 2047, r - max = 36, 6 bits
	return (((y) << 11) | (x));
}
__host__ __device__ int INT_TO_X(unsigned int v) {
	return (v)&((1 << 11) - 1);
}
__host__ __device__ int INT_TO_Y(unsigned int v) {
	return (v >> 11)&((1 << 11) - 1);
}

__host__ __device__ int cuMax(int a, int b) {
	if (a > b) {
		return a;
	}
	else {
		return b;
	}
}
__host__ __device__ int cuMin(int a, int b) {
	if (a < b) {
		return a;
	}
	else {
		return b;
	}
}

__device__ float MycuRand(hiprandState &state) {//random number in cuda, between 0 and 1
	
	 return hiprand_uniform(&state);

}
__device__ void InitcuRand(hiprandState &state) {//random number in cuda, between 0 and 1
	
	int i = threadIdx.x + blockIdx.x * blockDim.x;

    hiprand_init(i, 0, 0, &state);

}

__host__ Mat reconstruct_avg(Mat a, Mat b, unsigned int * ann, int patch_w) {

	Mat c;
	a.copyTo(c);
	for (int ay = 0; ay < a.rows; ay++) {
		for (int ax = 0; ax < a.cols; ax++)
		{
		
			float point_num = 0, *dist_tmp;
			
			dist_tmp = new float[3];

			for (int dc = 0; dc < 3; dc++)
			{
				dist_tmp[dc] = 0;
			}

			for (int dx = -patch_w / 2; dx <= patch_w / 2; dx++) {
				for (int dy = -patch_w / 2; dy <=patch_w / 2; dy++)
				{

					if ((ax + dx) < a.cols && (ax + dx) >= 0 && (ay + dy) < a.rows && (ay + dy) >= 0)
					{

						unsigned int vp = ann[(ay + dy)*a.cols + ax + dx];
						int xp = INT_TO_X(vp), yp = INT_TO_Y(vp);

						if ((xp - dx) < b.cols && (xp - dx) >= 0 && (yp - dy) < b.rows && (yp - dy) >= 0)//a patch that contain this pixel
						{

							for (int dc = 0; dc < 3; dc++)
							{
								dist_tmp[dc] += b.at<Vec3b>(yp - dy, xp - dx).val[dc];
							}

							point_num++;
						}
					}

				}

			}

			for (int dc = 0; dc < 3; dc++)
			{
				c.at<Vec3b>(ay, ax).val[dc] = dist_tmp[dc]/point_num;
			}
			
			delete[] dist_tmp;
		}
	}
	return c;
}

__host__ Mat reconstruct_dflow(Mat a, Mat b, unsigned int * ann, int patch_w) {
	Mat flow;
	a.copyTo(flow);
	for (int ay = 0; ay < a.rows; ay++) {
		for (int ax = 0; ax < a.cols; ax++)
		{
			unsigned int v = ann[ay*a.cols + ax];
			int xbest = INT_TO_X(v);
			int ybest = INT_TO_Y(v);
			flow.at<Vec3b>(ay, ax).val[0] = (uchar)(255 * ((float)(ax - xbest + b.cols - 1) / (2 * b.cols)));
			flow.at<Vec3b>(ay, ax).val[2] = (uchar)(0);
			flow.at<Vec3b>(ay, ax).val[1] = (uchar)(255 * ((float)(ay - ybest + b.rows - 1) / (2 * b.rows)));
		}
	}
	return flow;
}

__host__ __device__ float dist_compute(float * a, float * b, float * a1, float * b1, int channels, int a_rows, int a_cols, int b_rows, int b_cols, int ax, int ay, int bx, int by, int patch_w, float cutoff = INT_MAX) {//this is the average number of all matched pixel
																																																		  //suppose patch_w is an odd number
	float pixel_sum = 0, pixel_no = 0, pixel_dist = 0;//number of pixels realy counted
	float pixel_sum1 = 0;
	int a_slice = a_rows*a_cols, b_slice = b_rows*b_cols;
	int a_pitch = a_cols, b_pitch = b_cols;
	float dp_tmp;

	for (int dy = -patch_w / 2; dy <= patch_w / 2; dy++) {
		for (int dx = -patch_w / 2; dx <= patch_w / 2; dx++) {

			if (
				(ay + dy) < a_rows && (ay + dy) >= 0 && (ax + dx) < a_cols && (ax + dx) >= 0
				&&
				(by + dy) < b_rows && (by + dy) >= 0 && (bx + dx) < b_cols && (bx + dx) >= 0
				)//the pixel in a should exist and pixel in b should exist
			{
				if (channels == 3)
				{
					for (int dc = 0; dc < channels; dc++)
					{
						dp_tmp = a[dc * a_slice + (ay + dy) * a_pitch + (ax + dx)] - b[dc * b_slice + (by + dy) * b_pitch + (bx + dx)];
						pixel_sum += dp_tmp * dp_tmp;

						// HMM@ HACk
						/*if (1)
						{
							dp_tmp = a1[dc * a_slice + (ay + dy) * a_pitch + (ax + dx)] - b1[dc * b_slice + (by + dy) * b_pitch + (bx + dx)];
							pixel_sum1 += dp_tmp * dp_tmp;
						}*/

					}
				}
				else
				{
					for (int dc = 0; dc < channels; dc++)
					{
						dp_tmp = a[dc * a_slice + (ay + dy) * a_pitch + (ax + dx)] * b[dc * b_slice + (by + dy) * b_pitch + (bx + dx)];
						pixel_sum -= dp_tmp;

						// HMM@HACK
						/*if (1)
						{
							dp_tmp = a1[dc * a_slice + (ay + dy) * a_pitch + (ax + dx)] * b1[dc * b_slice + (by + dy) * b_pitch + (bx + dx)];
							pixel_sum1 -= dp_tmp;
						}*/
					}
				}
				

				pixel_no += 1;
			}
		}

	}

	// HMM@HACK
	if (pixel_no == 0)
	{
		pixel_dist = 0;
	}
	else
	{
		pixel_dist = (pixel_sum + pixel_sum1) / pixel_no;
	}
	if (pixel_dist >= cutoff) { return cutoff; }
	else {
		return pixel_dist;
	}
}

__host__ __device__ float dist(float * a, float * b, float *a1, float *b1, int channels, int a_rows, int a_cols, int b_rows, int b_cols, int ax, int ay, int xp, int yp, int patch_w, float cutoff = INT_MAX) {

	return dist_compute(a, b, a1, b1,  channels, a_rows, a_cols, b_rows, b_cols, ax, ay, xp, yp, patch_w, cutoff);

}

__device__ void improve_guess(float * a, float * b, float *a1, float *b1, int channels, int a_rows, int a_cols, int b_rows, int b_cols, int ax, int ay, int &xbest, int &ybest, float &dbest, int xp, int yp, int patch_w, float rr) {
	float d;
	d = dist(a, b, a1, b1, channels, a_rows, a_cols, b_rows, b_cols, ax, ay, xp, yp, patch_w, dbest);
	if (d + rr < dbest) {
		xbest = xp;
		ybest = yp;
		dbest = d;
	}
}


__global__ void initialAnn_kernel(unsigned int * ann, int * params) {

	//just use 7 of 9 parameters
	int ah = params[1];
	int aw = params[2];


	int ax = blockIdx.x*blockDim.x + threadIdx.x;
	int ay = blockIdx.y*blockDim.y + threadIdx.y;

	if (ax < aw && ay < ah) {
		int bx = ax;
		int by = ay;
		ann[ay*aw + ax] = XY_TO_INT(bx, by);
	}
}

__global__ void upSample_kernel(unsigned int * ann, unsigned int * ann_tmp,int * params, int aw_half,int ah_half) {

	int ax = blockIdx.x*blockDim.x + threadIdx.x;
	int ay = blockIdx.y*blockDim.y + threadIdx.y;

	
	int ah = params[1];
	int aw = params[2];
	int bh = params[3];
	int bw = params[4];
	
	
	float aw_ratio = (float)aw / (float)aw_half;
	float ah_ratio = (float)ah / (float)ah_half;
	int ax_half = (ax+0.5) / aw_ratio;
	int ay_half = (ay+0.5) / ah_ratio;
	ax_half = clamp(ax_half, aw_half - 1, 0);
	ay_half = clamp(ay_half, ah_half - 1, 0);
	

	if (ax < aw&&ay < ah) {

		unsigned int v_half = ann[ay_half*aw_half + ax_half];
		int bx_half = INT_TO_X(v_half);
		int by_half = INT_TO_Y(v_half);

		int bx = ax + (bx_half - ax_half)*aw_ratio + 0.5;
		int by = ay + (by_half - ay_half)*ah_ratio + 0.5;

		bx = clamp(bx, bw-1, 0);
		by = clamp(by, bh-1, 0);

		ann_tmp[ay*aw + ax] = XY_TO_INT(bx, by);
	}

}

__global__ void patchmatch(float * a, float * b, float *a1, float *b1, unsigned int *ann, float *annd, int * params) {

	int ax = blockIdx.x*blockDim.x + threadIdx.x;
	int ay = blockIdx.y*blockDim.y + threadIdx.y;

	//assign params
	int ch = params[0];
	int a_rows = params[1];
	int a_cols = params[2];
	int b_rows = params[3];
	int b_cols = params[4];
	int patch_w = params[5];
	int pm_iters = params[6];
	int rs_max = params[7];


	if (ax < a_cols && ay < a_rows) {
	
		// for random number
		hiprandState state;
		InitcuRand(state);

		unsigned int v, vp;

		int xp, yp, xbest, ybest;

		int xmin, xmax, ymin, ymax;

		float dbest;
		v = ann[ay*a_cols + ax];
		xbest = INT_TO_X(v), ybest = INT_TO_Y(v);			
		annd[ay*a_cols + ax] = dist(a, b, a1, b1,  ch, a_rows, a_cols, b_rows, b_cols, ax, ay, xbest, ybest, patch_w);

		for (int iter = 0; iter < pm_iters; iter++) {

			/* Current (best) guess. */
			v = ann[ay*a_cols + ax];
			xbest = INT_TO_X(v), ybest = INT_TO_Y(v);			
			dbest = annd[ay*a_cols + ax];

			/* In each iteration, improve the NNF, by jumping flooding. */
			for (int jump = 8; jump > 0; jump /= 2) {

				/* Propagation: Improve current guess by trying instead correspondences from left, right, up and downs. */
				if ((ax - jump) < a_cols && (ax - jump) >= 0)//left
				{
					vp = ann[ay*a_cols + ax - jump];//the pixel coordinates in image b

					xp = INT_TO_X(vp) + jump, yp = INT_TO_Y(vp);//the propagated match from vp, the center of the patch, which should be in the image

					if (yp >= 0 && yp < b_rows && xp >= 0 && xp < b_cols)
					{
						//improve guess
						improve_guess(a, b, a1, b1, ch, a_rows, a_cols, b_rows, b_cols, ax, ay, xbest, ybest, dbest, xp, yp, patch_w, 0);
						ann[ay*a_cols + ax] = XY_TO_INT(xbest, ybest);
						annd[ay*a_cols + ax] = dbest;
					}
				}
				
				if ((ax + jump) < a_cols)//right
				{
					vp = ann[ay*a_cols + ax + jump];//the pixel coordinates in image b

					xp = INT_TO_X(vp) - jump, yp = INT_TO_Y(vp);

					if (yp >= 0 && yp < b_rows && xp >= 0 && xp < b_cols)
					{
						//improve guess
						improve_guess(a, b, a1, b1, ch, a_rows, a_cols, b_rows, b_cols, ax, ay, xbest, ybest, dbest, xp, yp, patch_w, 0);
						ann[ay*a_cols + ax] = XY_TO_INT(xbest, ybest);
						annd[ay*a_cols + ax] = dbest;
					}
				}

				if ((ay - jump) < a_rows && (ay - jump) >= 0)//up
				{
					vp = ann[(ay - jump)*a_cols + ax];//the pixel coordinates in image b
					xp = INT_TO_X(vp), yp = INT_TO_Y(vp) + jump;

					if (yp >= 0 && yp < b_rows && xp >= 0 && xp < b_cols)
					{

						//improve guess
						improve_guess(a, b, a1, b1, ch, a_rows, a_cols, b_rows, b_cols, ax, ay, xbest, ybest, dbest, xp, yp, patch_w, 0);
						ann[ay*a_cols + ax] = XY_TO_INT(xbest, ybest);
						annd[ay*a_cols + ax] = dbest;
					}
				}

				if ((ay + jump) < a_rows)//down
				{
					vp = ann[(ay + jump)*a_cols + ax];//the pixel coordinates in image b	
					xp = INT_TO_X(vp), yp = INT_TO_Y(vp) - jump;

					if (yp >= 0 && yp < b_rows && xp >= 0 && xp < b_cols)
					{
						//improve guess
						improve_guess(a, b, a1, b1,  ch, a_rows, a_cols, b_rows, b_cols, ax, ay, xbest, ybest, dbest, xp, yp, patch_w, 0);
						ann[ay*a_cols + ax] = XY_TO_INT(xbest, ybest);
						annd[ay*a_cols + ax] = dbest;
					}
				}

			}

			/* Random search: Improve current guess by searching in boxes of exponentially decreasing size around the current best guess. */
			int rs_start = rs_max;
			if (rs_start > cuMax(b_cols, b_rows)) {
				rs_start = cuMax(b_cols, b_rows);
			}
			for (int mag = rs_start; mag >= 1; mag /= 2) {
				/* Sampling window */
				xmin = cuMax(xbest - mag, 0), xmax = cuMin(xbest + mag + 1, b_cols);
				ymin = cuMax(ybest - mag, 0), ymax = cuMin(ybest + mag + 1, b_rows);
				xp = xmin + (int)(MycuRand(state)*(xmax - xmin)) % (xmax - xmin);
				yp = ymin + (int)(MycuRand(state)*(ymax - ymin)) % (ymax - ymin);

				//improve guess
				improve_guess(a, b, a1, b1,  ch, a_rows, a_cols, b_rows, b_cols, ax, ay, xbest, ybest, dbest, xp, yp, patch_w, FLT_MIN);

			}

			ann[ay*a_cols + ax] = XY_TO_INT(xbest, ybest);
			annd[ay*a_cols + ax] = dbest;
			__syncthreads();
		}
	}
}

__global__ void blend(float *cmap, float* oldd, float* newd, float weight,int * params)
{
	int ax = blockIdx.x*blockDim.x + threadIdx.x;
	int ay = blockIdx.y*blockDim.y + threadIdx.y;

	int ch = params[0];
	int ah = params[1];
	int aw = params[2];
	
	int slice_a = ah * aw;
	int pitch_a = aw;
	
	// HMM@ HACK
	float thre = 0.05;
	
	if (ax < aw&& ay < ah)
	{
		float fa = cmap[ay*pitch_a + ax];
		
		if (fa < thre)
			fa = 0.0f;

		else fa = weight;

		for (int i = 0; i < ch; i++)
		{
			
			newd[i*slice_a + ay*pitch_a + ax] = oldd[i*slice_a + ay*pitch_a + ax]* fa + newd[i*slice_a + ay*pitch_a + ax] * (1.0-fa);
		}
	}
}

// ********** VOTE ***********

__global__ void avg_vote(unsigned int * ann, float * pb, float * pc, int * params) {

	int ax = blockIdx.x*blockDim.x + threadIdx.x;
	int ay = blockIdx.y*blockDim.y + threadIdx.y;

	int ch = params[0];
	int ah = params[1];
	int aw = params[2];
	int bh = params[3];
	int bw = params[4];
	int patch_w = params[5];

	int slice_a = ah * aw;
	int pitch_a = aw;
	int slice_b = bh * bw;
	int pitch_b = bw;

	int count = 0;

	if (ax < aw&&ay < ah)
	{

		//set zero for all the channels at (ax,ay)
		for (int i = 0; i < ch; i++)
		{
			pc[i*slice_a + ay*pitch_a + ax] = 0;

		}

		//count the sum of all the possible value of (ax,ay)
		for (int dx = -patch_w / 2; dx <= patch_w / 2; dx++) {
			for (int dy = -patch_w / 2; dy <= patch_w / 2; dy++)
			{

				if ((ax + dx) < aw && (ax + dx) >= 0 && (ay + dy) < ah && (ay + dy) >= 0)
				{
					unsigned int vp = ann[(ay + dy)*aw + ax + dx];
					
					int xp = INT_TO_X(vp);
					int yp = INT_TO_Y(vp);

					if ((xp - dx) < bw && (xp - dx) >= 0 && (yp - dy) < bh && (yp - dy) >= 0)
					{
						count++;
						for (int dc = 0; dc < ch; dc++)
						{
							pc[dc*slice_a + ay*pitch_a + ax] += pb[dc*slice_b + (yp - dy)*pitch_b + xp - dx];
						}
					}
				}

			}
		}

		//count average value
		for (int i = 0; i < ch; i++)
		{
			pc[i*slice_a + ay*pitch_a + ax] /= count;
		}

	}
}

__global__ void compute_dist(unsigned int* ann, float* annd, float* fa, float* fb, int* params, int full_w, int full_h)
{
	int full_x = blockIdx.x*blockDim.x + threadIdx.x;
	int full_y = blockIdx.y*blockDim.y + threadIdx.y;

	int ch = params[0];
	int ah = params[1];
	int aw = params[2];
	int bh = params[3];
	int bw = params[4];

	int slice_a = ah * aw;
	int slice_b = bh * bw;

	float ratio_x = (float)(aw - 1) / (float)(full_w - 1);
	float ratio_y = (float)(ah - 1) / (float)(full_h - 1);

	if (full_x < full_w && full_y < full_h)
	{
		float nx = full_x * ratio_x;
		float ny = full_y * ratio_y;

		int nax0 = min(int(nx), aw - 1);
		int nay0 = min(int(ny), ah - 1);
		int nax1 = min(nax0 + 1, aw - 1);
		int nay1 = min(nay0 + 1, ah - 1);

		float wax1 = nx - nax0;
		float way1 = ny - nay0;
		float wax0 = 1 - wax1;
		float way0 = 1 - way1;

		int aid00 = nay0 * aw + nax0;
		int aid01 = nay0 * aw + nax1;
		int aid10 = nay1 * aw + nax0;
		int aid11 = nay1 * aw + nax1;

		unsigned int v = ann[full_y * full_w + full_x];
		int bx = INT_TO_X(v);
		int by = INT_TO_Y(v);
		nx = bx * ratio_x;
		ny = by * ratio_y;
		int nbx0 = min(int(nx), bw - 1);
		int nby0 = min(int(ny), bh - 1);
		int nbx1 = min(nbx0 + 1, bw - 1);
		int nby1 = min(nby0 + 1, bh - 1);

		float wbx1 = nx - nbx0;
		float wby1 = ny - nby0;
		float wbx0 = 1 - wbx1;
		float wby0 = 1 - wby1;

		int bid00 = nby0 * bw + nbx0;
		int bid01 = nby0 * bw + nbx1;
		int bid10 = nby1 * bw + nbx0;
		int bid11 = nby1 * bw + nbx1;

		int id = full_y * full_w + full_x;

		annd[id] = 0;
		for (int c = 0; c < ch; ++c)
		{
			int bid = c * slice_b;
			int aid = c * slice_a;
			annd[id] -=
				(wax0 * way0 * fa[aid + aid00] +
				wax0 * way1 * fa[aid + aid10] +
				wax1 * way0 * fa[aid + aid01] +
				wax1 * way1 * fa[aid + aid11]) *
				(wbx0 * wby0 * fb[bid + bid00] +
				wbx0 * wby1 * fb[bid + bid10] +
				wbx1 * wby0 * fb[bid + bid01] +
				wbx1 * wby1 * fb[bid + bid11]);
		}
	}
}

__global__ void reverse_dist(unsigned int* ann, float* annd, float* bnnd, int aw, int ah, int bw, int bh) {

	int ax = blockIdx.x*blockDim.x + threadIdx.x;
	int ay = blockIdx.y*blockDim.y + threadIdx.y;

	if (ax < aw&&ay < ah) {

		unsigned int v = ann[ay * aw + ax];
		int bx = INT_TO_X(v);
		int by = INT_TO_Y(v);
		if (bx < bw && by < bh)
		{
			annd[ay * aw + ax] = bnnd[by * bw + bx];
		}
		else
		{
			annd[ay * aw + ax] = 0;
		}
	}
}

__global__ void compute_dist_norm(unsigned int* ann, float* annd, float* fa, float* fb, int* params, int full_w, int full_h, int scale)
{
	int full_x = blockIdx.x*blockDim.x + threadIdx.x;
	int full_y = blockIdx.y*blockDim.y + threadIdx.y;

	int ch = params[0];
	int ah = params[1];
	int aw = params[2];
	int bh = params[3];
	int bw = params[4];

	int slice_a = ah * aw;
	int slice_b = bh * bw;

	float padding_shift = scale * 0.5f;
	if (scale == 1)
	{
		padding_shift = 0;
	}

	if (full_x < full_w && full_y < full_h)
	{
		int id = full_y * full_w + full_x;

		float nx = (full_x + padding_shift) / (float)scale;
		float ny = (full_y + padding_shift) / (float)scale;

		int nax0 = min(int(nx), aw - 1);
		int nay0 = min(int(ny), ah - 1);
		int nax1 = min(nax0 + 1, aw - 1);
		int nay1 = min(nay0 + 1, ah - 1);

		float wax1 = nx - nax0;
		float way1 = ny - nay0;
		float wax0 = 1 - wax1;
		float way0 = 1 - way1;

		int aid00 = nay0 * aw + nax0;
		int aid01 = nay0 * aw + nax1;
		int aid10 = nay1 * aw + nax0;
		int aid11 = nay1 * aw + nax1;

		unsigned int v = ann[id];
		int bx = INT_TO_X(v);
		int by = INT_TO_Y(v);

		nx = (bx + padding_shift) / (float)scale;
		ny = (by + padding_shift) / (float)scale;

		int nbx0 = min(int(nx), bw - 1);
		int nby0 = min(int(ny), bh - 1);
		int nbx1 = min(nbx0 + 1, bw - 1);
		int nby1 = min(nby0 + 1, bh - 1);

		float wbx1 = nx - nbx0;
		float wby1 = ny - nby0;
		float wbx0 = 1 - wbx1;
		float wby0 = 1 - wby1;

		int bid00 = nby0 * bw + nbx0;
		int bid01 = nby0 * bw + nbx1;
		int bid10 = nby1 * bw + nbx0;
		int bid11 = nby1 * bw + nbx1;

		annd[id] = 0;
		float asum = 0;
		float bsum = 0;
		for (int c = 0; c < ch; ++c)
		{
			int bid = c * slice_b;
			int aid = c * slice_a;
			float af =
				(wax0 * way0 * fa[aid + aid00] +
				wax0 * way1 * fa[aid + aid10] +
				wax1 * way0 * fa[aid + aid01] +
				wax1 * way1 * fa[aid + aid11]);
			float bf =
				(wbx0 * wby0 * fb[bid + bid00] +
				wbx0 * wby1 * fb[bid + bid10] +
				wbx1 * wby0 * fb[bid + bid01] +
				wbx1 * wby1 * fb[bid + bid11]);

			annd[id] -= af * bf;
			asum += af * af;
			bsum += bf * bf;
		}
		asum = max(sqrt(asum), 0.00000001f);
		bsum = max(sqrt(bsum), 0.00000001f);
		annd[id] /= asum * bsum;
	}
}

__global__ void compute_dist_inplace(float* annd, float* fa, float* fb, int* params, int full_w, int full_h, int scale)
{
	int full_x = blockIdx.x*blockDim.x + threadIdx.x;
	int full_y = blockIdx.y*blockDim.y + threadIdx.y;

	int ch = params[0];
	int ah = params[1];
	int aw = params[2];
	int bh = params[3];
	int bw = params[4];

	int slice_a = ah * aw;
	int slice_b = bh * bw;

	float padding_shift = scale * 0.5f;
	if (scale == 1)
	{
		padding_shift = 0;
	}

	if (full_x < full_w && full_y < full_h)
	{
		int id = full_y * full_w + full_x;

		float nx = (full_x + padding_shift) / (float)scale;
		float ny = (full_y + padding_shift) / (float)scale;

		int nax0 = min(int(nx), aw - 1);
		int nay0 = min(int(ny), ah - 1);
		int nax1 = min(nax0 + 1, aw - 1);
		int nay1 = min(nay0 + 1, ah - 1);

		float wax1 = nx - nax0;
		float way1 = ny - nay0;
		float wax0 = 1 - wax1;
		float way0 = 1 - way1;

		int aid00 = nay0 * aw + nax0;
		int aid01 = nay0 * aw + nax1;
		int aid10 = nay1 * aw + nax0;
		int aid11 = nay1 * aw + nax1;

		annd[id] = 0;
		float asum = 0;
		float bsum = 0;
		for (int c = 0; c < ch; ++c)
		{
			//int bid = c * slice_b;
			int aid = c * slice_a;
			float af =
				(wax0 * way0 * fa[aid + aid00] +
				wax0 * way1 * fa[aid + aid10] +
				wax1 * way0 * fa[aid + aid01] +
				wax1 * way1 * fa[aid + aid11]);
			float bf =
				(wax0 * way0 * fb[aid + aid00] +
				wax0 * way1 * fb[aid + aid10] +
				wax1 * way0 * fb[aid + aid01] +
				wax1 * way1 * fb[aid + aid11]);

			annd[id] -= af * bf;
			asum += af * af;
			bsum += bf * bf;
		}
		asum = max(sqrt(asum), 0.00000001f);
		bsum = max(sqrt(bsum), 0.00000001f);
		annd[id] /= asum * bsum;
	}
}

__global__ void compute_l2dist_inplace(float* annd, float* fa, float* fb, int* params, int full_w, int full_h, int scale)
{
	int full_x = blockIdx.x*blockDim.x + threadIdx.x;
	int full_y = blockIdx.y*blockDim.y + threadIdx.y;

	int ch = params[0];
	int ah = params[1];
	int aw = params[2];
	int bh = params[3];
	int bw = params[4];

	int slice_a = ah * aw;
	int slice_b = bh * bw;

	float padding_shift = scale * 0.5f;
	if (scale == 1)
	{
		padding_shift = 0;
	}

	if (full_x < full_w && full_y < full_h)
	{
		int id = full_y * full_w + full_x;

		float nx = (full_x + padding_shift) / (float)scale;
		float ny = (full_y + padding_shift) / (float)scale;

		int nax0 = min(int(nx), aw - 1);
		int nay0 = min(int(ny), ah - 1);
		int nax1 = min(nax0 + 1, aw - 1);
		int nay1 = min(nay0 + 1, ah - 1);

		float wax1 = nx - nax0;
		float way1 = ny - nay0;
		float wax0 = 1 - wax1;
		float way0 = 1 - way1;

		int aid00 = nay0 * aw + nax0;
		int aid01 = nay0 * aw + nax1;
		int aid10 = nay1 * aw + nax0;
		int aid11 = nay1 * aw + nax1;

		annd[id] = 0;
		//float asum = 0;
		//float bsum = 0;
		for (int c = 0; c < ch; ++c)
		{
			//int bid = c * slice_b;
			int aid = c * slice_a;
			float af =
				(wax0 * way0 * fa[aid + aid00] +
				wax0 * way1 * fa[aid + aid10] +
				wax1 * way0 * fa[aid + aid01] +
				wax1 * way1 * fa[aid + aid11]);
			float bf =
				(wax0 * way0 * fb[aid + aid00] +
				wax0 * way1 * fb[aid + aid10] +
				wax1 * way0 * fb[aid + aid01] +
				wax1 * way1 * fb[aid + aid11]);

			annd[id] += (af - bf) * (af - bf);
		}
	}
}

__global__ void compute_l1dist_inplace(float* annd, float* fa, float* fb, int* params, int full_w, int full_h, int scale)

{
	int full_x = blockIdx.x*blockDim.x + threadIdx.x;
	int full_y = blockIdx.y*blockDim.y + threadIdx.y;

	int ch = params[0];
	int ah = params[1];
	int aw = params[2];
	int bh = params[3];
	int bw = params[4];

	int slice_a = ah * aw;
	int slice_b = bh * bw;

	float padding_shift = scale * 0.5f;
	if (scale == 1)
	{
		padding_shift = 0;
	}

	if (full_x < full_w && full_y < full_h)
	{
		int id = full_y * full_w + full_x;

		float nx = (full_x + padding_shift) / (float)scale;
		float ny = (full_y + padding_shift) / (float)scale;

		int nax0 = min(int(nx), aw - 1);
		int nay0 = min(int(ny), ah - 1);
		int nax1 = min(nax0 + 1, aw - 1);
		int nay1 = min(nay0 + 1, ah - 1);

		float wax1 = nx - nax0;
		float way1 = ny - nay0;
		float wax0 = 1 - wax1;
		float way0 = 1 - way1;

		int aid00 = nay0 * aw + nax0;
		int aid01 = nay0 * aw + nax1;
		int aid10 = nay1 * aw + nax0;
		int aid11 = nay1 * aw + nax1;

		annd[id] = 0;

		for (int c = 0; c < ch; ++c)
		{
			int aid = c * slice_a;
			float af =
				(wax0 * way0 * fa[aid + aid00] +
				wax0 * way1 * fa[aid + aid10] +
				wax1 * way0 * fa[aid + aid01] +
				wax1 * way1 * fa[aid + aid11]);
			float bf =
				(wax0 * way0 * fb[aid + aid00] +
				wax0 * way1 * fb[aid + aid10] +
				wax1 * way0 * fb[aid + aid01] +
				wax1 * way1 * fb[aid + aid11]);

			annd[id] += abs(af - bf);
		}
	}
}

__global__ void convert_float2bgr(float* annd, unsigned char* bgr, int w, int h)
{
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;

	if (x < w && y < h)
	{
		int id = y * w + x;
		int err = max(min((annd[id] + 1.f), 1.f), 0.f) * 255.f;

		bgr[id] = err;
	}
}

__global__ void convert_float2bgr(float* annd, unsigned char* bgr, int w, int h, float minval, float maxval)
{
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;

	if (x < w && y < h)
	{
		int id = y * w + x;
		int err = max(min((annd[id] - minval) / (maxval - minval), 1.f), 0.f) * 255.f;

		bgr[id] = err;
	}
}

__global__ void reverse_flow(unsigned int* ann, unsigned int* bnn, unsigned int* rann, int ah, int aw, int bh, int bw) {

	int ax = blockIdx.x*blockDim.x + threadIdx.x;
	int ay = blockIdx.y*blockDim.y + threadIdx.y;

	if (ax < aw && ay < ah) 
	{
		unsigned int v = ann[ay * aw + ax];
		int bx = INT_TO_X(v);
		int by = INT_TO_Y(v);

		if (bx < bw && by < bh)
		{
			rann[ay * aw + ax] = bnn[by * bw + bx];
		}
		else
		{
			rann[ay * aw + ax] = XY_TO_INT(ax, ay);
		}
	}
}